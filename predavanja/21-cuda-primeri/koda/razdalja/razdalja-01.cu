#include "hip/hip_runtime.h"
// računanje razdalje med vektorjema
// 		argumenti: število blokov, število niti in dolžina vektorjev
// 		elementi vektorjev so inicializirani naključno
// osnovna rešitev: razdaljo izračunamo na gostitelju
// primer argumentov: srun --reservation=psistemi --partition=gpu --gpus=1 razdalja 0 1024 67108864

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

__global__ void vectorDistance(float *c, const float *a, const float *b, int len) {
	// računanje razlike
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	while (gid < len) {
		c[gid] = a[gid] - b[gid];
		gid += gridDim.x * blockDim.x;
	}
}

int main(int argc, char **argv) {
	// preberemo argumente iz ukazne vrstice
	int numBlocks = 0;
	int numThreads = 0;
	int vectorLength = 0;
	if (argc == 4) {
		numBlocks = atoi(argv[1]);
		numThreads = atoi(argv[2]);
		vectorLength = atoi(argv[3]);
	}
	if (numBlocks < 0 || numThreads <= 0 || ceil(log2(numThreads)) != floor(log2(numThreads)) || vectorLength <= 0) {
		printf("usage:\n\t%s <number of blocks> <number of threads (power of 2)> <vector length>\n", argv[0]);
		exit(EXIT_FAILURE);
	}

	// določimo potrebno število blokov niti
	if (numBlocks == 0)
		numBlocks = (vectorLength - 1) / numThreads + 1;

	// rezerviramo pomnilnik na gostitelju
	float *hc = (float *)malloc(vectorLength * sizeof(float));
	float *ha = (float *)malloc(vectorLength * sizeof(float));
	float *hb = (float *)malloc(vectorLength * sizeof(float));

	// rezerviramo pomnilnik na napravi
	float *dc, *da, *db;
	checkCudaErrors(hipMalloc((void**)&dc, vectorLength * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&da, vectorLength * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&db, vectorLength * sizeof(float)));

	// nastavimo vrednosti vektorjev a in b na gostitelju
	srand(time(NULL));
	for (int i = 0; i < vectorLength; i++) {
		ha[i] = (float)rand()/RAND_MAX;
		hb[i] = (float)rand()/RAND_MAX;
	}

	// merjenje časa na napravi - začetek
	struct timespec startDevice, stopDevice;
	clock_gettime(CLOCK_MONOTONIC, &startDevice);

	// prenesemo vektorja a in b iz gostitelja na napravo
	checkCudaErrors(hipMemcpy(da, ha, vectorLength * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(db, hb, vectorLength * sizeof(float), hipMemcpyHostToDevice));

	// zaženemo kodo na napravi
	dim3 gridSize(numBlocks, 1, 1);
	dim3 blockSize(numThreads, 1, 1);
	vectorDistance<<<gridSize, blockSize>>>(dc, da, db, vectorLength);
	checkCudaErrors(hipGetLastError());

	// počakamo, da vse niti na napravi zaključijo
	checkCudaErrors(hipDeviceSynchronize());

	// vektor c prekopiramo iz naprave na gostitelja
	checkCudaErrors(hipMemcpy(hc, dc, vectorLength * sizeof(float), hipMemcpyDeviceToHost));

	// dokončamo izračun razdalje za napravo
	double distDevice = 0.0;
	for (int i = 0; i < vectorLength; i++)
		distDevice += hc[i] * hc[i];
	distDevice = sqrt(distDevice);

	// merjenje časa na napravi - konec
	clock_gettime(CLOCK_MONOTONIC, &stopDevice);
	double timeDevice = (stopDevice.tv_sec - startDevice.tv_sec) * 1e3 + (stopDevice.tv_nsec - startDevice.tv_nsec) / 1e6;

	// izračunamo razdaljo in izmerimo čas na gostitelju
	struct timespec startHost, stopHost;
	clock_gettime(CLOCK_MONOTONIC, &startHost);
	double distHost = 0.0;
	float diff;
	for (int i = 0; i < vectorLength; i++) {
		diff = ha[i] - hb[i];
		distHost += diff * diff;
	}
	distHost = sqrt(distHost);
	clock_gettime(CLOCK_MONOTONIC, &stopHost);
	double timeHost = (stopHost.tv_sec - startHost.tv_sec) * 1e3 + (stopHost.tv_nsec - startHost.tv_nsec) / 1e6;

	// rezultata izpišemo
	printf("naprava:      %lf (%lf ms)\ngostitelj:    %lf (%lf ms)\nnapaka (rel): %e\n", distDevice, timeDevice, distHost, timeHost, fabs(distDevice/distHost-1));

	// sprostimo pomnilnik na napravi
	checkCudaErrors(hipFree(dc));
	checkCudaErrors(hipFree(da));
	checkCudaErrors(hipFree(db));

	// sprostimo pomnilnik na gostitelju
	free(hc);
	free(ha);
	free(hb);

	return 0;
}
