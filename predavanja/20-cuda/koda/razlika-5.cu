#include "hip/hip_runtime.h"
// računanje razlike vektorjev
// 		argumenti: število blokov, število niti in dolžina vektorjev
// 		elementi vektorjev so inicializirani naključno
// dobra rešitev z računanjem števila blokov

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

__global__ void vectorSubtract(float *c, const float *a, const float *b, int len) {
	// določimo globalni indeks elementov
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	// če je niti manj kot je dolžina vektorjev, morajo nekatere narediti več elementov
	while (gid < len) {
		c[gid] = a[gid] - b[gid];
		gid += gridDim.x * blockDim.x;
	}
}

int main(int argc, char **argv) {
	// preberemo argumente iz ukazne vrstice
	int numBlocks = 0;
	int numThreads = 0;
	int vectorLength = 0;
	if (argc == 4) {
		numBlocks = atoi(argv[1]);
		numThreads = atoi(argv[2]);
		vectorLength = atoi(argv[3]);
	}
	if (numBlocks < 0 || numThreads <= 0 || vectorLength <= 0) {
		printf("usage:\n\t%s <number of blocks> <number of threads> <vector length>\n", argv[0]);
		exit(EXIT_FAILURE);
	}

	// določimo potrebno število blokov niti
	if (numBlocks == 0)
		numBlocks = (vectorLength - 1) / numThreads + 1;

	// rezerviramo pomnilnik na gostitelju
	float *hc = (float *)malloc(vectorLength * sizeof(float));
	float *ha = (float *)malloc(vectorLength * sizeof(float));
	float *hb = (float *)malloc(vectorLength * sizeof(float));

	// rezerviramo pomnilnik na napravi
	float *da, *db, *dc;
	checkCudaErrors(hipMalloc((void**)&da, vectorLength * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&db, vectorLength * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dc, vectorLength * sizeof(float)));

	// nastavimo vrednosti vektorjev a in b na gostitelju
	srand(time(NULL));
	for (int i = 0; i < vectorLength; i++) {
		ha[i] = (float)rand()/RAND_MAX;
		hb[i] = (float)rand()/RAND_MAX;
	}

	// prenesemo vektorja a in b iz gostitelja na napravo
	checkCudaErrors(hipMemcpy(da, ha, vectorLength * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(db, hb, vectorLength * sizeof(float), hipMemcpyHostToDevice));

	// zaženemo kodo na napravi
	dim3 gridSize(numBlocks, 1, 1);
	dim3 blockSize(numThreads, 1, 1);
	vectorSubtract<<<gridSize, blockSize>>>(dc, da, db, vectorLength);
	checkCudaErrors(hipGetLastError());

	// počakamo, da vse niti na napravi zaključijo
	checkCudaErrors(hipDeviceSynchronize());

	// vektor c prekopiramo iz naprave na gostitelja
	checkCudaErrors(hipMemcpy(hc, dc, vectorLength * sizeof(float), hipMemcpyDeviceToHost));

	// preverimo rezultat
	int ok = 1;
	for (int i = 0; i < vectorLength; i++)
		ok &= (ha[i] - hb[i]) == hc[i];
	printf("Result is %s.\n", ok == 1 ? "correct": "wrong");

	// sprostimo pomnilnik na napravi
	checkCudaErrors(hipFree(dc));
	checkCudaErrors(hipFree(da));
	checkCudaErrors(hipFree(db));

	// sprostimo pomnilnik na gostitelju
	free(hc);
	free(ha);
	free(hb);

	return 0;
}
