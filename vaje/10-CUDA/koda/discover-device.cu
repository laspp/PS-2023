#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"

int main(int argc, char **argv) {
  
  // Get number of GPUs
  int deviceCount = 0;
  hipError_t error = hipGetDeviceCount(&deviceCount);

  if (error != hipSuccess) {
    printf("hipGetDeviceCount error %d\n-> %s\n", error, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }

  // Get device propreties and print 
  for (int dev = 0; dev < deviceCount; dev++) {
    struct hipDeviceProp_t prop;
    int value;
    printf("\n==========  cudaDeviceGetProperties ============  \n");
    checkCudaErrors(hipGetDeviceProperties(&prop, dev));
    printf("\nDevice %d: \"%s\"\n", dev, prop.name);
    printf("  GPU Clock Rate (MHz):                          %d\n", prop.clockRate/1000);
    printf("  Memory Clock Rate (MHz):                       %d\n", prop.memoryClockRate/1000);
    printf("  Memory Bus Width (bits):                       %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s):                  %.2f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  CUDA Cores/MP:                                 %d\n", _ConvertSMVer2Cores(prop.major, prop.minor));
    printf("  CUDA Cores:                                    %d\n", _ConvertSMVer2Cores(prop.major, prop.minor) *
           prop.multiProcessorCount);
    printf("  Total amount of global memory:                 %.0f GB\n", prop.totalGlobalMem / 1073741824.0f);
    printf("  Total amount of shared memory per block:       %zu kB\n",
           prop.sharedMemPerBlock/1024);
    printf("  Total number of registers available per block: %d\n",
           prop.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           prop.warpSize);
    printf("  Maximum number of threads per block:           %d\n",
           prop.maxThreadsPerBlock);
    printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1],
           prop.maxThreadsDim[2]);
    printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1],
           prop.maxGridSize[2]);

    printf("\n\n==========  hipDeviceGetAttribute ============  \n");
    printf("\nDevice %d: \"%s\"\n", dev, prop.name);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxThreadsPerBlock, dev);
    printf("  Max number of threads per block:              %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlockDimX, dev);
    printf("  Max block dimension X:                        %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlockDimY, dev);
    printf("  Max block dimension Y:                        %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlockDimZ, dev);
    printf("  Max block dimension Z:                        %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxGridDimX, dev);
    printf("  Max grid dimension X:                         %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxGridDimY, dev);
    printf("  Max grid dimension Y:                         %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxGridDimZ, dev);
    printf("  Max grid dimension Z:                         %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxSharedMemoryPerBlock, dev);
    printf("  Max shared memory per block:                  %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeWarpSize, dev);
    printf("  Warp size:                                    %d\n",
           value);      
    hipDeviceGetAttribute (&value, hipDeviceAttributeClockRate, dev);
    printf("  Peak clock frequency in kilohertz:            %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMemoryClockRate, dev);
    printf("  Peak memory clock frequency in kilohertz:     %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMemoryBusWidth, dev);
    printf("  Global memory bus width in bits:              %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeL2CacheSize, dev);
    printf("  Size of L2 cache in bytes:                    %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
    printf("  Maximum resident threads per SM:              %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeComputeCapabilityMajor, dev);
    printf("  Major compute capability version number:      %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeComputeCapabilityMinor, dev);
    printf("  Minor compute capability version number:      %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, dev);
    printf("  Max shared memory per SM in bytes:            %d\n",
           value);
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev);
    printf("  Max number of 32-bit registers per SM:        %d\n",
           value);  
    hipDeviceGetAttribute (&value, hipDeviceAttributeSharedMemPerBlockOptin, dev);
    printf("  Max per block shared mem size on the device:  %d\n",
           value);  
    hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlocksPerMultiprocessor, dev);
    printf("  Max thread blocks that can reside on a SM:    %d\n",
           value);  
  }
}