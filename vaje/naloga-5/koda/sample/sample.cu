#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 1

__global__ void do_nothing(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height)
{
    printf("DID NOTHING\n");
}

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        printf("USAGE: sample input_image output_image\n");
        exit(EXIT_FAILURE);
    }
    
    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);

    // Load image from file and allocate space for the output image
    int width, height, cpp;
    unsigned char *h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);
    cpp = COLOR_CHANNELS;

    if (h_imageIn == NULL)
    {
        printf("Error reading loading image %s!\n", szImage_in_name);
        exit(EXIT_FAILURE);
    }
    printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);
    const size_t datasize = width * height * cpp * sizeof(unsigned char);
    unsigned char *h_imageOut = (unsigned char *)malloc(datasize);

    // Kot preizkus samo kopiramo vhodno sliko v izhodno
    memcpy(h_imageOut,h_imageIn,datasize);

    // Nastavimo organizacijo niti v 2D
    dim3 blockSize(1, 1);
    dim3 gridSize(1, 1);

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;

    // Rezervacija pomnilnika na napravi
    checkCudaErrors(hipMalloc(&d_imageIn, datasize));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));

    // Uporabimo dogodke CUDA za merjenje casa
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Zazenemo scepec
    hipEventRecord(start);
    do_nothing<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height);
    getLastCudaError("do_nothing() execution failed\n");
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    // Izpisemo cas
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution time is: %0.3f milliseconds \n", milliseconds);

    // Zapisemo izhodno sliko v datoteko
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");
    char *FileType = NULL;
    while (token != NULL)
    {
        FileType = token;
        token = strtok(NULL, ".");
    }

    if (!strcmp(FileType, "png"))
        stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
    else if (!strcmp(FileType, "jpg"))
        stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
    else if (!strcmp(FileType, "bmp"))
        stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
    else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    // Sprostimo pomnilnik na napravi
    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));

    // Pocistimo dogodke
	hipEventDestroy(start);
	hipEventDestroy(stop);
    
    // Sprostimo pomnilnik na gostitelju
    free(h_imageIn);
    free(h_imageOut);

    return 0;
}
